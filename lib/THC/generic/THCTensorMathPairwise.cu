
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathPairwise.cu"
#else

THC_API void
THCTensor_(add)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorAddConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorAddConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(sub)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorSubConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorSubConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(mul)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorMulConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorMulConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(div)(THCState* state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(value != ScalarConvert<int, real>::to(0), 3, "divide by zero");

  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorDivConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorDivConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(tril)(THCState *state, THCTensor *self_, THCTensor *src_, long k)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCTensor *src = src_;
  if (self_ == src_)
    src = THCTensor_(newContiguous)(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  real *start = THCTensor_(data)(state, src) + src->storageOffset;

  TensorTriOp<real, 0> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCTensor_(freeCopyTo)(state, src, src_);

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(triu)(THCState *state, THCTensor *self_, THCTensor *src_, long k)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCTensor *src = src_;
  if (self_ == src_)
    src = THCTensor_(newContiguous)(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  real *start = THCTensor_(data)(state, src) + src->storageOffset;

  TensorTriOp<real, 1> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCTensor_(freeCopyTo)(state, src, src_);

  THCudaCheck(cudaGetLastError());
}

#endif
